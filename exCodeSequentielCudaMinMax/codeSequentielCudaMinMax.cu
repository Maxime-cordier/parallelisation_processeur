/*==============================================================================*/
/* Programme 	: CodeSequentiel.c												*/
/* Auteur 	: Daniel CHILLET													*/
/* Date 	: Decembre 2021														*/
/* 																				*/
/*==============================================================================*/


#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <time.h>

#define MAX_CHAINE 100
#define MAX_HOSTS 100

#define CALLOC(ptr, nr, type) 		if (!(ptr = (type *) calloc((size_t)(nr), sizeof(type)))) {		\
						printf("Erreur lors de l'allocation memoire \n") ; 		\
						exit (-1);							\
					} 

#define FOPEN(fich,fichier,sens) 	if ((fich=fopen(fichier,sens)) == NULL) { 				\
						printf("Probleme d'ouverture du fichier %s\n",fichier);		\
						exit(-1);							\
					} 
				
#define MIN(a, b) 	(a < b ? a : b)
#define MAX(a, b) 	(a > b ? a : b)

#define MAX_VALEUR 	255
#define MIN_VALEUR 	0

#define NBPOINTSPARLIGNES 15

#define false 0
#define true 1
#define boolean int

#define InitClock    struct timespec start, stop
#define ClockStart   clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &start)
#define ClockEnd   clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &stop)
#define BILLION  1000000000L
#define ClockMesureSec "%2.9f s\n",(( stop.tv_sec - start.tv_sec )+ (stop.tv_nsec - start.tv_nsec )/(double)BILLION) 

#define BLOCKSIZE 10

#define DEBUG (0)
#define TPSCALCUL (1)

__global__ void rehaussement_contraste(int *image, int *res, float etalement, int min, long N) {
	long i = (long)blockIdx.x * (long)blockDim.x + (long)threadIdx.x;
	if (i < N) {
		res[i] = (image[i] - min) * etalement;
	}
}

__global__ void calcul_min_max(int *imageMin, int *imageMax, int tailleImage, long N) {
	long i = (long)blockIdx.x * (long)blockDim.x + (long)threadIdx.x;
	
	if(i < N) {
		int decalage = tailleImage/2;
		if (tailleImage%2 != 0){
			decalage++;
		}
		imageMin[i] = MIN(imageMin[i], imageMin[i+decalage]);
		imageMax[i] = MAX(imageMax[i], imageMax[i+decalage]);
	}
}

int main(int argc, char **argv) {
	/*========================================================================*/
	/* Declaration de variables et allocation memoire */
	/*========================================================================*/

	int i, n;
	
	int LE_MIN = MAX_VALEUR;
	int LE_MAX = MIN_VALEUR;
	
	float ETALEMENT = 0.0;
	
	int *image;
	int *resultat;
	int X, Y, cpt;
	int TailleImage;
	
	int P;
	
	FILE *Src, *Dst;

	char SrcFile[MAX_CHAINE];
	char DstFile[MAX_CHAINE+4];
	char ligne[MAX_CHAINE];

	boolean inverse = false;
	
	char *Chemin;
	

InitClock;

	/*========================================================================*/
	/* Recuperation des parametres						*/
	/*========================================================================*/


	if (argc != 2){
		printf("Syntaxe : CodeSequentiel image.pgm \n");
		exit(-1);
	}
	sscanf(argv[1],"%s", SrcFile);
	
	sprintf(DstFile,"%s.new",SrcFile);
	
	/*========================================================================*/
	/* Recuperation de l'endroit ou l'on travail				*/
	/*========================================================================*/

	CALLOC(Chemin, MAX_CHAINE, char);
	Chemin = getenv("PWD");
	if DEBUG printf("Repertoire de travail : %s \n\n",Chemin);

	/*========================================================================*/
	/* Ouverture des fichiers						*/
	/*========================================================================*/

	if DEBUG printf("Operations sur les fichiers\n");

	FOPEN(Src, SrcFile, "r");
	if DEBUG printf("\t Fichier source ouvert (%s) \n",SrcFile);
		
	FOPEN(Dst, DstFile, "w");
	if DEBUG printf("\t Fichier destination ouvert (%s) \n",DstFile);
	
	/*========================================================================*/
	/* On effectue la lecture du fichier source */
	/*========================================================================*/
	
	if DEBUG printf("\t Lecture entete du fichier source ");
	
	for (i = 0 ; i < 2 ; i++) {
		fgets(ligne, MAX_CHAINE, Src);	
		fprintf(Dst,"%s", ligne);
	}	

	fscanf(Src," %d %d\n",&X, &Y);
	fprintf(Dst," %d %d\n", X, Y);
	
	fgets(ligne, MAX_CHAINE, Src);	/* Lecture du 255 	*/
	fprintf(Dst,"%s", ligne);
	
	if DEBUG printf(": OK \n");
	
	/*========================================================================*/
	/* Allocation m#include <immintrin.h>emoire pour l'image source et l'image resultat 		*/
	/*========================================================================*/
	
	TailleImage = X * Y;

	CALLOC(image, X*Y, int);
	CALLOC(resultat, X*Y, int);

	if DEBUG printf("\t\t Initialisation de l'image [%d ; %d] : Ok \n", X, Y);
	
	/*========================================================================*/
	/* Lecture du fichier pour remplir l'image source 			*/
	/*========================================================================*/
	
	cpt = 0;
	while (! feof(Src)) {
		n = fscanf(Src,"%d",&P);

		image[cpt] = P;
		cpt ++;
		if (n == EOF || (cpt == X*Y)) {
			break;
		}
	}


	fclose(Src);
	if DEBUG printf("\t Lecture du fichier image : Ok \n\n");

	int TailleImageTmp = TailleImage;
	int *cudaImageMin;
	int *cudaImageMax;
	int size = TailleImage*sizeof(int);
	
	long dimBlock = BLOCKSIZE;
	long dimGrid;
	
	if (hipMalloc((void **)&cudaImageMin, size) == hipErrorOutOfMemory) {
		printf("Allocation memoire qui pose probleme (cudaVec) \n");
	}
	if (hipMalloc((void **)&cudaImageMax, size) == hipErrorOutOfMemory) {
		printf("Allocation memoire qui pose probleme (cudaVec) \n");
	}
	
	hipMemcpy(&cudaImageMin[0], &image[0], size, hipMemcpyHostToDevice);
	hipMemcpy(&cudaImageMax[0], &image[0], size, hipMemcpyHostToDevice);


	while (TailleImageTmp != 1) {

		int nbThreadNecessaires = TailleImageTmp/2;
		if(TailleImageTmp%2 != 0) {
			nbThreadNecessaires++;
		}

		dimGrid = (TailleImageTmp/dimBlock)/2 + 1;
		calcul_min_max<<< dimGrid, dimBlock >>>(cudaImageMin, cudaImageMax, TailleImageTmp, nbThreadNecessaires);
		TailleImageTmp = TailleImageTmp/2;
	}

	hipMemcpy(&LE_MIN, &cudaImageMin[0], sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(&LE_MAX, &cudaImageMax[0], sizeof(int), hipMemcpyDeviceToHost);

	if DEBUG printf("\t Min %d ; Max %d \n\n", LE_MIN, LE_MAX);

	/*========================================================================*/
	/* Calcul du facteur d'etalement					*/
	/*========================================================================*/
	
	if (inverse) {
		ETALEMENT = 0.2;	
	} else {
		ETALEMENT = (float)(MAX_VALEUR - MIN_VALEUR) / (float)(LE_MAX - LE_MIN);	
	}
	
	/*========================================================================*/
	/* Calcul de cahque nouvelle valeur de pixel							*/
	/*========================================================================*/

	int *cuda_image;
	int *cuda_resultat;

	size = TailleImage * sizeof(int);
	
	if (hipMalloc((void **)&cuda_image, size) == hipErrorOutOfMemory) {
		printf("Allocation memoire qui pose probleme (cudaVec) \n");
	}
	if (hipMalloc((void **)&cuda_resultat, size)  == hipErrorOutOfMemory) {
		printf("Allocation memoire qui pose probleme (cudaRes) \n");
	}

	dimBlock = BLOCKSIZE;
	dimGrid = TailleImage/BLOCKSIZE;
	if ((TailleImage % BLOCKSIZE) != 0) {
		dimGrid++;
	}
	
	int res = hipMemcpy(&cuda_image[0], &image[0], size, hipMemcpyHostToDevice);
ClockStart;
	rehaussement_contraste<<<dimGrid, dimBlock>>>(cuda_image, cuda_resultat, ETALEMENT, LE_MIN, TailleImage);
ClockEnd;
	hipMemcpy(&resultat[0], &cuda_resultat[0], size, hipMemcpyDeviceToHost);

if TPSCALCUL printf(ClockMesureSec);

	/*========================================================================*/
	/* Sauvegarde de l'image dans le fichier resultat			*/
	/*========================================================================*/
	
	n = 0;

	for (i=0; i<X*Y ; i++) {
		fprintf(Dst,"%3d ", resultat[i]);
		n++;
		if (n == NBPOINTSPARLIGNES) {
			n = 0;
			fprintf(Dst, "\n");
		}
	}
				
	fprintf(Dst,"\n");
	fclose(Dst);
	
	printf("\n");

	/*========================================================================*/
	/* Fin du programme principal	*/
	/*========================================================================*/
	
	exit(0); 
	
}
