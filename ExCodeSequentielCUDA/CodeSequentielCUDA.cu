/*==============================================================================*/
/* Programme 	: CodeSequentiel.c												*/
/* Auteur 	: Daniel CHILLET													*/
/* Date 	: Decembre 2021														*/
/* 																				*/
/*==============================================================================*/


#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>


#define MAX_CHAINE 100
#define MAX_HOSTS 100

#define CALLOC(ptr, nr, type) 		if (!(ptr = (type *) calloc((size_t)(nr), sizeof(type)))) {		\
						printf("Erreur lors de l'allocation memoire \n") ; 		\
						exit (-1);							\
					} 

#define FOPEN(fich,fichier,sens) 	if ((fich=fopen(fichier,sens)) == NULL) { 				\
						printf("Probleme d'ouverture du fichier %s\n",fichier);		\
						exit(-1);							\
					} 
				
#define MIN(a, b) 	(a < b ? a : b)
#define MAX(a, b) 	(a > b ? a : b)

#define MAX_VALEUR 	255
#define MIN_VALEUR 	0

#define NBPOINTSPARLIGNES 15

#define false 0
#define true 1
#define boolean int

#include <time.h>

#define InitClock    struct timespec start, stop
#define ClockStart   clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &start)
#define ClockEnd   clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &stop)
#define BILLION  1000000000L
#define ClockMesureSec "%2.9f s\n",(( stop.tv_sec - start.tv_sec )+ (stop.tv_nsec - start.tv_nsec )/(double)BILLION) 


#define BLOCKSIZE 1024

#define DEBUG (0)
#define TPSCALCUL (1)

__global__ void rehaussement_contraste(int *image, int *res, float etalement, int min, long N) {
	long i = (long)blockIdx.x * (long)blockDim.x + (long)threadIdx.x;
	if (i < N) {
		res[i] = (image[i] - min) * etalement;
	}
}

int main(int argc, char **argv) {
	/*========================================================================*/
	/* Declaration de variables et allocation memoire */
	/*========================================================================*/

	int i, n;
	
	int LE_MIN = MAX_VALEUR;
	int LE_MAX = MIN_VALEUR;
	
	float ETALEMENT = 0.0;
	
	int *image;
	int *resultat;
	int X, Y, cpt;
	int TailleImage;
	
	int P;
	
	FILE *Src, *Dst;

	char SrcFile[MAX_CHAINE];
	char DstFile[MAX_CHAINE+4];
	char ligne[MAX_CHAINE];

	boolean inverse = false;
	
	char *Chemin;
	

InitClock;

	/*========================================================================*/
	/* Recuperation des parametres						*/
	/*========================================================================*/


	if (argc != 2){
		printf("Syntaxe : CodeSequentiel image.pgm \n");
		exit(-1);
	}
	sscanf(argv[1],"%s", SrcFile);
	
	sprintf(DstFile,"%s.new",SrcFile);
	
	/*========================================================================*/
	/* Recuperation de l'endroit ou l'on travail				*/
	/*========================================================================*/

	CALLOC(Chemin, MAX_CHAINE, char);
	Chemin = getenv("PWD");
	if DEBUG printf("Repertoire de travail : %s \n\n",Chemin);

	/*========================================================================*/
	/* Ouverture des fichiers						*/
	/*========================================================================*/

	if DEBUG printf("Operations sur les fichiers\n");

	FOPEN(Src, SrcFile, "r");
	if DEBUG printf("\t Fichier source ouvert (%s) \n",SrcFile);
		
	FOPEN(Dst, DstFile, "w");
	if DEBUG printf("\t Fichier destination ouvert (%s) \n",DstFile);
	
	/*========================================================================*/
	/* On effectue la lecture du fichier source */
	/*========================================================================*/
	
	if DEBUG printf("\t Lecture entete du fichier source ");
	
	for (i = 0 ; i < 2 ; i++) {
		fgets(ligne, MAX_CHAINE, Src);	
		fprintf(Dst,"%s", ligne);
	}	

	fscanf(Src," %d %d\n",&X, &Y);
	fprintf(Dst," %d %d\n", X, Y);
	
	fgets(ligne, MAX_CHAINE, Src);	/* Lecture du 255 	*/
	fprintf(Dst,"%s", ligne);
	
	if DEBUG printf(": OK \n");
	
	/*========================================================================*/
	/* Allocation m#include <immintrin.h>emoire pour l'image source et l'image resultat 		*/
	/*========================================================================*/
	
	TailleImage = X * Y;

	CALLOC(image, X*Y, int);
	CALLOC(resultat, X*Y, int);

/*
	CALLOC(image, Y+1, int *);
	CALLOC(resultat, Y+1, int *);
	for (i=0;i<Y;i++) {
		CALLOC(image[i], X+1, int);
		CALLOC(resultat[i], X+1, int);
		for (j=0;j<X;j++) {
			image[i][j] = 0;
			resultat[i][j] = 0;
		}
	}
*/
	if DEBUG printf("\t\t Initialisation de l'image [%d ; %d] : Ok \n", X, Y);
			
	
	/*x = 0;
	y = 0;*/
	cpt = 0;
	
	//lignes = 0;
	
	/*========================================================================*/
	/* Lecture du fichier pour remplir l'image source 			*/
	/*========================================================================*/
	
	while (! feof(Src)) {
		n = fscanf(Src,"%d",&P);

		image[cpt] = P;
		/*image[y][x] = P;*/
		
		//x ++;
		cpt ++;

		if (n == EOF || (cpt == X*Y)) {
			break;
		}


		/*if (x == X) {
			x = 0 ;
			y++;
		}*/
	}


	fclose(Src);
	if DEBUG printf("\t Lecture du fichier image : Ok \n\n");

	for (i=0; i<X*Y ; i++) {
		LE_MIN = MIN(LE_MIN, image[i]);
		LE_MAX = MAX(LE_MAX, image[i]);
	}

	printf("le min : %d \n", LE_MIN);

	/*
	for (i=0;i<Y;i++) {
		for (j=0;j<X;j++) {
			LE_MIN = MIN(LE_MIN, image[i][j]);
			LE_MAX = MAX(LE_MAX, image[i][j]);
		}
	}*/

	if DEBUG printf("\t Min %d ; Max %d \n\n", LE_MIN, LE_MAX);


	/*========================================================================*/
	/* Calcul du facteur d'etalement					*/
	/*========================================================================*/
	
	if (inverse) {
		ETALEMENT = 0.2;	
	} else {
		ETALEMENT = (float)(MAX_VALEUR - MIN_VALEUR) / (float)(LE_MAX - LE_MIN);	
	}
	
	/*========================================================================*/
	/* Calcul de cahque nouvelle valeur de pixel							*/
	/*========================================================================*/

	int *cuda_image;
	int *cuda_resultat;

	int size = TailleImage * sizeof(int);
	
	if (hipMalloc((void **)&cuda_image, size) == hipErrorOutOfMemory) {
		printf("Allocation memoire qui pose probleme (cudaVec) \n");
	}
	if (hipMalloc((void **)&cuda_resultat, size)  == hipErrorOutOfMemory) {
		printf("Allocation memoire qui pose probleme (cudaRes) \n");
	}

	long dimBlock = BLOCKSIZE;
	long dimGrid = TailleImage/BLOCKSIZE;
	if ((TailleImage % BLOCKSIZE) != 0) {
		dimGrid++;
	}
	
	int res = hipMemcpy(&cuda_image[0], &image[0], size, hipMemcpyHostToDevice);
ClockStart;
	rehaussement_contraste<<<dimGrid, dimBlock>>>(cuda_image, cuda_resultat, ETALEMENT, LE_MIN, TailleImage);
ClockEnd;
	hipMemcpy(&resultat[0], &cuda_resultat[0], size, hipMemcpyDeviceToHost);

if TPSCALCUL printf(ClockMesureSec);

	/*========================================================================*/
	/* Sauvegarde de l'image dans le fichier resultat			*/
	/*========================================================================*/
	
	n = 0;

	for (i=0; i<X*Y ; i++) {
		fprintf(Dst,"%3d ", resultat[i]);
		n++;
		if (n == NBPOINTSPARLIGNES) {
			n = 0;
			fprintf(Dst, "\n");
		}
	}

	/*
	for (i = 0 ; i < Y ; i++) {
		for (j = 0 ; j < X ; j++) {
			
			fprintf(Dst,"%3d ",resultat[i][j]);
			n++;
			if (n == NBPOINTSPARLIGNES) {
				n = 0;
				fprintf(Dst, "\n");
			}
		}
	}*/
				
	fprintf(Dst,"\n");
	fclose(Dst);
	
	printf("\n");

	/*========================================================================*/
	/* Fin du programme principal	*/
	/*========================================================================*/
	
	exit(0); 
	
}
