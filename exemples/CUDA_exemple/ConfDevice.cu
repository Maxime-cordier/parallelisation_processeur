#include <stdio.h> 

int main() {

  FILE * inputFile = fopen("/users/imr/rheritea/Documents/Processeur_multimedia/parallelisation_processeur/gpuinfo_barn-e-01", "w");
  if ( inputFile == NULL ) {
    fprintf( stderr, "Cannot open file \n");
    exit( 0 );
  }

  int nDevices;

  hipGetDeviceCount(&nDevices);
  for (int i = 0; i < nDevices; i++) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, i);
    fprintf( inputFile, "Device Number: %d\n", i);
    fprintf( inputFile, "  Device name: %s\n", prop.name);
    fprintf( inputFile, "  Memory Clock Rate (KHz): %d\n",
           prop.memoryClockRate);
    fprintf( inputFile, "  Global Memory (bytes): %ld\n",
           prop.totalGlobalMem);
    fprintf( inputFile, "  Memory Bus Width (bits): %d\n",
           prop.memoryBusWidth);
    fprintf( inputFile, "  Peak Memory Bandwidth (GB/s): %f\n",
           2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
    fprintf( inputFile, "   Max Thread per block : %d \n",prop.maxThreadsPerBlock);
    fprintf( inputFile, "   Multiproc count : %d \n",prop.multiProcessorCount);
    fprintf( inputFile, "   Max Grid size : %d %d %d \n",prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
    fprintf( inputFile, "   Max thread dim : %d %d %d \n",prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
    fprintf( inputFile, "   Registres per block : %d \n",prop.regsPerBlock);
    fprintf( inputFile, "\n");
  }
} 
